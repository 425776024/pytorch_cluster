#include "hip/hip_runtime.h"
#include "fps_cuda.h"

#include "utils.cuh"

inline torch::Tensor get_dist(torch::Tensor x, int64_t idx) {
  return (x - x[idx]).norm(2, 1);
}

torch::Tensor fps_cuda(torch::Tensor src, torch::Tensor ptr, double ratio,
                       bool random_start) {

  CHECK_CUDA(src);
  CHECK_CUDA(ptr);
  CHECK_INPUT(ptr.dim() == 1);
  AT_ASSERTM(ratio > 0 and ratio < 1, "Invalid input");

  src = src.view({src.size(0), -1}).contiguous();
  ptr = ptr.contiguous();
  auto batch_size = ptr.size(0) - 1;

  auto deg = ptr.narrow(0, 1, batch_size) - ptr.narrow(0, 0, batch_size);
  auto out_ptr = deg.toType(torch::kFloat) * (float)ratio;
  out_ptr = out_ptr.ceil().toType(torch::kLong).cumsum(0);
  out_ptr = torch::cat({torch.zeros(1, ptr.options()), out_ptr}, 0);

  torch::Tensor start;
  if (random_start) {
    start = at::rand(batch_size, src.options());
    start = (start * deg.toType(torch::kFloat)).toType(torch::kLong);
  } else {
    start = torch::zeros(batch_size, ptr.options());
  }

  auto out = torch::empty(out_ptr[-1].data_ptr<int64_t>()[0], ptr.options());

  auto ptr_data = ptr.data_ptr<int64_t>();
  auto out_ptr_data = out_ptr.data_ptr<int64_t>();
  auto out_data = out.data_ptr<int64_t>();

  return out;
}
