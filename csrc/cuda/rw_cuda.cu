#include "hip/hip_runtime.h"
#include "rw_cuda.h"

#include <ATen/cuda/HIPContext.h>

#include "utils.cuh"

#define THREADS 1024
#define BLOCKS(N) (N + THREADS - 1) / THREADS

__global__ void uniform_random_walk_kernel(const int64_t *rowptr,
                                           const int64_t *col,
                                           const int64_t *start,
                                           const float *rand, int64_t *n_out,
                                           int64_t *e_out, int64_t walk_length,
                                           int64_t numel) {
  const int64_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (thread_idx < numel) {
    int64_t n_cur = start[thread_idx], e_cur, row_start, row_end, rnd;

    n_out[thread_idx] = n_cur;

    for (int64_t l = 0; l < walk_length; l++) {
      row_start = rowptr[n_cur], row_end = rowptr[n_cur + 1];
      if (row_end - row_start == 0) {
        e_cur = -1;
      } else {
        rnd = int64_t(rand[l * numel + thread_idx] * (row_end - row_start));
        e_cur = row_start + rnd;
        n_cur = col[e_cur];
      }
      n_out[(l + 1) * numel + thread_idx] = n_cur;
      e_out[l * numel + thread_idx] = e_cur;
    }
  }
}

torch::Tensor random_walk_cuda(torch::Tensor rowptr, torch::Tensor col,
                               torch::Tensor start, int64_t walk_length,
                               double p, double q) {
  CHECK_CUDA(rowptr);
  CHECK_CUDA(col);
  CHECK_CUDA(start);
  hipSetDevice(rowptr.get_device());

  CHECK_INPUT(rowptr.dim() == 1);
  CHECK_INPUT(col.dim() == 1);
  CHECK_INPUT(start.dim() == 1);

  auto rand = torch::rand({start.size(0), walk_length},
                          start.options().dtype(torch::kFloat));

  auto n_out = torch::empty({walk_length + 1, start.size(0)}, start.options());
  auto e_out = torch::empty({walk_length, start.size(0)}, start.options());

  auto stream = at::cuda::getCurrentCUDAStream();
  uniform_random_walk_kernel<<<BLOCKS(start.numel()), THREADS, 0, stream>>>(
      rowptr.data_ptr<int64_t>(), col.data_ptr<int64_t>(),
      start.data_ptr<int64_t>(), rand.data_ptr<float>(),
      n_out.data_ptr<int64_t>(), e_out.data_ptr<int64_t>(), walk_length,
      start.numel());

  return n_out.t().contiguous();
}
